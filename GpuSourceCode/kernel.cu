#include "hip/hip_runtime.h"
#include <chrono>  // for high_resolution_clock
#include "hip/hip_runtime.h"
#include ""
#include <iostream> 
#include <thrust/sort.h>
#include <stdio.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <algorithm>
#include <time.h>
#include <filesystem>
#if defined(_MSC_VER)
#include <direct.h>
#define getcwd _getcwd
#elif defined(__GNUC__)
#include <unistd.h>
#endif
#include <iomanip>
#include <sstream>
#include <fstream>
#include <sys/stat.h>
#include <string>

/*
rozmiary siatek
100x100
4    4
25	 25

1000x1000
25   25
40	 40

10000x10000
40   40
250	 250
*/

enum DivideType {
	LesserThanZero,
	HaveZero,
	GreaterThanZero
};

class DirectedOrderNumber {

public:
	double infimum, supremum;
	bool zeroFlag;
	__device__  __host__
	DirectedOrderNumber()
	{
		infimum = 0.0;
		supremum = 0.0;
		zeroFlag = false;
	}
	__device__  __host__
	DirectedOrderNumber(const DirectedOrderNumber& that)
	{
		infimum = that.infimum;
		supremum = that.supremum;
		zeroFlag = false;
	}
	__device__ __host__
	DirectedOrderNumber(double newInfimum, double newSupremum)
	{
		infimum = double(newInfimum);
		supremum = double(newSupremum);
		zeroFlag = false;
	}
	__device__ __host__
	DirectedOrderNumber(DirectedOrderNumber* orderNumber)
	{
		infimum = orderNumber->infimum;
		supremum = orderNumber->supremum;
		zeroFlag = orderNumber->zeroFlag;
	}
	__device__ __host__
	void SetValues(double newInf, double newSup)
	{
		infimum = newInf;
		supremum = newSup;
		zeroFlag = false;
	}
	__device__
		double GetRealNumber()
	{
		return (infimum + supremum) / 2.0;
	}
	__device__
		double GetAbsoulteRealNumber()
	{
		double result = (infimum + supremum) / 2.0;

		if (result < 0)
			return -1 * result;

		else
			return result;
	}
	__device__
		DivideType GetDivideType()
	{
		if (infimum > 0.0 && supremum > 0.0)
		{
			return GreaterThanZero;
		}

		else if (infimum < 0.0 && supremum < 0.0)
		{
			return LesserThanZero;
		}
		else
		{
			return HaveZero;
		}
	}
};

__device__
void AddDirectedOrderNumber(DirectedOrderNumber* firstValue, DirectedOrderNumber* secondValue, DirectedOrderNumber* result)
{
	result->SetValues(firstValue->infimum + secondValue->infimum, firstValue->supremum + secondValue->supremum);
}
__device__
void SubstractDirectedOrderNumber(DirectedOrderNumber* firstValue, DirectedOrderNumber* secondValue, DirectedOrderNumber* result)
{
	result->SetValues(firstValue->infimum - secondValue->infimum, firstValue->supremum - secondValue->supremum);
}

__device__
void SortMultipleArray(double* array)
{
	double tmp = 0.0;
	for (int i = 0; i < 4; i++)
	{
		for (int j = 1; j < 4 - i; j++) {
			if (array[j - 1] > array[j])
			{
				tmp = array[j - 1];
				array[j - 1] = array[j];
				array[j] = tmp;
			}
		}
	}
}

__device__
void MultipleDirectedOrderNumber(DirectedOrderNumber* firstNumber, DirectedOrderNumber* secondNumber, DirectedOrderNumber* result) {

	double* multipleResult;
	multipleResult = new double[4];
	multipleResult[0] = (firstNumber->infimum * secondNumber->infimum);
	multipleResult[1] = (firstNumber->infimum * secondNumber->supremum);
	multipleResult[2] = (firstNumber->supremum * secondNumber->supremum);
	multipleResult[3] = (firstNumber->supremum * secondNumber->infimum);

	SortMultipleArray(multipleResult);
	result->infimum = multipleResult[0];
	result->supremum = multipleResult[3];
	delete[](multipleResult);
}

__device__
void DivideDirectedOrderNumber(DirectedOrderNumber* firstNumber, DirectedOrderNumber* secondNumber, DirectedOrderNumber* result) {

	if (firstNumber->GetDivideType() == GreaterThanZero && secondNumber->GetDivideType() == GreaterThanZero)
	{
		result->infimum = firstNumber->infimum / secondNumber->infimum;
		result->supremum = firstNumber->supremum / secondNumber->supremum;
	}

	else if (firstNumber->GetDivideType() == LesserThanZero && secondNumber->GetDivideType() == LesserThanZero)
	{
		result->supremum = firstNumber->infimum / secondNumber->infimum;
		result->infimum = firstNumber->supremum / secondNumber->supremum;
	}
	else if (firstNumber->GetDivideType() == GreaterThanZero && secondNumber->GetDivideType() == LesserThanZero)
	{
		result->infimum = firstNumber->supremum / secondNumber->infimum;
		result->supremum = firstNumber->infimum / secondNumber->supremum;
	}
	else if (firstNumber->GetDivideType() == LesserThanZero && secondNumber->GetDivideType() == GreaterThanZero)
	{
		result->infimum = firstNumber->infimum / secondNumber->supremum;
		result->supremum = firstNumber->supremum / secondNumber->infimum;
	}

	else if (firstNumber->GetDivideType() == HaveZero && secondNumber->GetDivideType() == GreaterThanZero)
	{
		result->infimum = firstNumber->infimum / secondNumber->supremum;
		result->supremum = firstNumber->supremum / secondNumber->supremum;
	}
	else if (firstNumber->GetDivideType() == HaveZero && secondNumber->GetDivideType() == LesserThanZero)
	{
		result->infimum = firstNumber->supremum / secondNumber->infimum;
		result->supremum = firstNumber->infimum / secondNumber->infimum;
	}
	else if (secondNumber->GetDivideType() == HaveZero)
	{
		secondNumber->infimum = (secondNumber->GetRealNumber() / 2.0) - 0.00000001;
		secondNumber->supremum = (secondNumber->GetRealNumber() / 2.0) + 0.00000001;
		DivideDirectedOrderNumber(firstNumber, secondNumber, result);
	}

}

__global__
void SetMnoznikMatrix(DirectedOrderNumber* matrix, DirectedOrderNumber* mnoznik, int size, int actuall_Collumn)
{	
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * size + column;
	DirectedOrderNumber minus(-1.0, -1.0);
	if (index < size * size)
	{
		mnoznik[index].infimum = 1.0;
		mnoznik[index].supremum = 1.0;
	}

	if (index == row * size + actuall_Collumn && index != row * (size + 1))
	{
		MultipleDirectedOrderNumber(&mnoznik[index], &minus, &mnoznik[index]);	
		DivideDirectedOrderNumber(&mnoznik[index], &matrix[(actuall_Collumn * size) + actuall_Collumn], &mnoznik[index]);
	}
	__syncthreads();
	if (index != row * size + actuall_Collumn)
	{
		mnoznik[index] = mnoznik[row * size + actuall_Collumn];
	}
}

__global__
void EleminatePart(DirectedOrderNumber* matrix, DirectedOrderNumber* cmatrix, DirectedOrderNumber* devBVector, int size, int actuall_Collumn)
{
	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int index = row * size + column;
	DirectedOrderNumber tmp(0.0, 0.0);
	DirectedOrderNumber tmpTwo(0.0, 0.0);
	if (row > actuall_Collumn)
	{	
		MultipleDirectedOrderNumber(&cmatrix[row * size + actuall_Collumn], &matrix[(actuall_Collumn * size) + column], &tmp);
		if (matrix[index].zeroFlag == false)
		{
			AddDirectedOrderNumber(&matrix[index], &tmp, &matrix[index]);
		}
	}
	__syncthreads();
	if (row == actuall_Collumn)
	{
		matrix[actuall_Collumn * size + row].zeroFlag = true;
	}
	if (index < size && index > actuall_Collumn)
	{
		MultipleDirectedOrderNumber(&cmatrix[column * size + actuall_Collumn], &devBVector[actuall_Collumn], &tmpTwo);
		AddDirectedOrderNumber(&devBVector[index], &tmpTwo, &devBVector[index]);
	}
	__syncthreads();

}



__global__
void SetReductionTable(DirectedOrderNumber* reductionVector, DirectedOrderNumber* matrix, DirectedOrderNumber* resultVector, int size, int actual_column)
{


	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int index = row * size + column;

	if (index > actual_column * size + row && index < size * size && row == actual_column && index != row * (size + 1))
	{
		MultipleDirectedOrderNumber(&matrix[actual_column * size + column], &resultVector[column], &reductionVector[column]);
	}

	else
	{
		reductionVector[index].infimum = 0.0;
		reductionVector[index].supremum = 0.0;
	}

}

__global__
void Reduction(DirectedOrderNumber* reductionVector, int size, DirectedOrderNumber* result)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	auto step_size = 1;
	int number_of_threads = blockDim.x;
	while (number_of_threads > 0)
	{
		if (index < number_of_threads)
		{
			const auto fst = index * step_size * 2;
			const auto snd = fst + step_size;
			AddDirectedOrderNumber(&reductionVector[fst], &reductionVector[snd], &reductionVector[fst]);
		}
		step_size <<= 1;
		number_of_threads >>= 1;
	}
	if (index == 0)
	{
		result->infimum = reductionVector[0].infimum;
		result->supremum = reductionVector[0].supremum;
	}
}

__global__
void GainResolveVector(DirectedOrderNumber* matrix, DirectedOrderNumber* resultVector, DirectedOrderNumber* bVector, DirectedOrderNumber* reductedValue, int size, int acutalColumn)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	DirectedOrderNumber sum = bVector[acutalColumn];
	SubstractDirectedOrderNumber(&sum, reductedValue, &sum);
	DivideDirectedOrderNumber(&sum, &matrix[acutalColumn * (size + 1)], &resultVector[acutalColumn]);
}

__global__
void SetResultVector(DirectedOrderNumber* resultVector, int size)
{
	int column = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int index = row * size + column;
	if (index < size)
	{
		resultVector[index].infimum = 1.0;
		resultVector[index].supremum = 1.0;
	}
}




DirectedOrderNumber *GaussElemination(DirectedOrderNumber* equation, DirectedOrderNumber* bvector, int size, int blocksize, int threadGridsize)
{
	DirectedOrderNumber* devVector, * devBValues, * mnoznikMatrix, * VectorResult;

	size_t matrixSize = size * size * sizeof(DirectedOrderNumber);
	size_t resultVectorSize = size * sizeof(DirectedOrderNumber);

	hipMalloc((void**)&devVector, matrixSize);
	hipMalloc((void**)&mnoznikMatrix, matrixSize);
	hipMalloc((void**)&VectorResult, resultVectorSize);
	hipMalloc((void**)&devBValues, resultVectorSize);
	hipEvent_t start, stop;
	hipEvent_t start2;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start2);
	hipEventRecord(start);
	hipMemcpy(devVector, equation, matrixSize, hipMemcpyHostToDevice);
	hipMemcpy(devBValues, bvector, resultVectorSize, hipMemcpyHostToDevice);

	dim3 blockGrid(blocksize, blocksize);
	dim3 thredGrid(threadGridsize, threadGridsize);
	dim3 RdimBlock(1, 1, 1);
	dim3 RdimGrid(size, 1, 1);
	SetResultVector << < blockGrid, thredGrid >> > (VectorResult, size);
	hipEventRecord(start2);
	for (int i = 0; i < size; i++)
	{
		SetMnoznikMatrix << <blockGrid, thredGrid >> > (devVector, mnoznikMatrix, size, i);
		EleminatePart << < blockGrid, thredGrid >> > (devVector, mnoznikMatrix, devBValues, size, i);
		std::cout << "Aktualny krok eliminacji: " << i << std::endl;
	}


	DirectedOrderNumber* minusVector;
	hipMalloc((void**)&minusVector, resultVectorSize);
	DirectedOrderNumber* reductionResult;
	hipMalloc((void**)&reductionResult, sizeof(DirectedOrderNumber));
	for (int i = size-1; i >= 0; i--)
	{
		SetReductionTable <<<blockGrid, thredGrid >>> (minusVector, devVector, VectorResult, size, i);
		Reduction << <RdimBlock, RdimGrid >> > (minusVector, size, reductionResult);
		GainResolveVector << <1, 1 >> > (devVector, VectorResult, devBValues, reductionResult, size, i);
		std::cout << "Aktualny krok uzyskania wektoru: " << i << std::endl;
	}
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0, milisecondstwo = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventElapsedTime(&milisecondstwo, start2, stop);
	std::cout << "Czas w milisekundach po stronie GPU:" << milliseconds << std::endl;
	std::cout << "Czas w milisekundach po stronie GPU bez alkoacji pamieci:" << milisecondstwo << std::endl;
	DirectedOrderNumber* hostResult;
	hostResult = new DirectedOrderNumber[size];
	hipMemcpy(hostResult, VectorResult, resultVectorSize, hipMemcpyDeviceToHost);

	hipFree(reductionResult);
	hipFree(minusVector);
	hipFree(devVector);
	hipFree(devBValues);
	hipFree(mnoznikMatrix);
	hipFree(VectorResult);

	return hostResult;
}



/*Koniec elimiancji Gaussa Funkcji*/
/*#####################################################################*/
/*Eliminiacja Gaussa-Jordana*/


__global__
void SetCMatirxJordan(DirectedOrderNumber* devMatrix, DirectedOrderNumber* cmatrix, int rowSize, int acutall_Column)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * rowSize + column;

	if (index == row * rowSize + acutall_Column)
	{
		DivideDirectedOrderNumber(&devMatrix[index], &devMatrix[(acutall_Column * rowSize) + acutall_Column], &cmatrix[index]);

	}

	if (index < rowSize * rowSize)
	{
		if (index != row * rowSize + acutall_Column)
		{
			cmatrix[index].infimum = cmatrix[row * rowSize + acutall_Column].infimum;
			cmatrix[index].supremum = cmatrix[row * rowSize + acutall_Column].supremum;
		}
	}
}


__global__
void EleminatePartGPUJordan(DirectedOrderNumber* devMatrix, DirectedOrderNumber* cmatrix, DirectedOrderNumber* bvector, int rowSize, int acutall_Column)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * rowSize + column;
	DirectedOrderNumber tmp2(-1.0, -1.0);
	DirectedOrderNumber tmp(0.0, 0.0);
	if (index < rowSize * rowSize)
	{
		if (row != acutall_Column)
		{
			if (devMatrix[index].zeroFlag == false)
			{
				MultipleDirectedOrderNumber(&cmatrix[row * rowSize + acutall_Column], &devMatrix[(acutall_Column * rowSize) + column], &tmp);
				SubstractDirectedOrderNumber(&devMatrix[index], &tmp, &devMatrix[index]);
			}	
		}
		if (index < rowSize && index != acutall_Column)
		{
			MultipleDirectedOrderNumber(&cmatrix[column * rowSize + acutall_Column], &bvector[acutall_Column], &tmp2);
			SubstractDirectedOrderNumber(&bvector[index], &tmp2, &bvector[index]);
		}
		__syncthreads();
		if (column*rowSize*row == acutall_Column)
		{
			devMatrix[index].zeroFlag = true;
		}

		}
		
		
		__syncthreads();
}

__global__
void SetResolveVectorJordan(DirectedOrderNumber* matrix, DirectedOrderNumber* vector, int size)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * size + column;

	if (index < size)
	{
		vector[index] = matrix[index * (size + 1)];
	}

}

__global__
void GetResovleVectorJordan(DirectedOrderNumber* diagonal, DirectedOrderNumber* bvector, DirectedOrderNumber* result, int size)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * size + column;

	if (index < size)
	{
		DivideDirectedOrderNumber(&bvector[index], &diagonal[index], &result[index]);
	}

}


__global__
void printMatrix(DirectedOrderNumber* x, int size)
{
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int column = (blockIdx.x * blockDim.x) + threadIdx.x;
	int index = row * size + column;

	if (index < size)
	{
		printf(" watke: %d", index);
		printf(" kontrola: %f",x[index].infimum);
	}

}

DirectedOrderNumber *GaussJordanElemination(DirectedOrderNumber * hmatrix, DirectedOrderNumber* hvectorb, int size, int blockGridSize, int ThreadSize)
{
	DirectedOrderNumber* dMatrix, * dBVector, *cMatrix, *dResultVector;
	size_t matrixSize = size * size * sizeof(DirectedOrderNumber);
	size_t resultVectorSize = size * sizeof(DirectedOrderNumber);
	hipEvent_t start, stop;
	hipEvent_t start2;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&start2);

	hipMalloc((void**)&dMatrix, matrixSize);
	hipMalloc((void**)&cMatrix, matrixSize);
	hipMalloc((void**)&dBVector, resultVectorSize);
	hipMalloc((void**)&dResultVector, resultVectorSize);
	hipEventRecord(start);
	hipMemcpy(dMatrix, hmatrix, matrixSize, hipMemcpyHostToDevice);
	hipMemcpy(dBVector, hvectorb, resultVectorSize, hipMemcpyHostToDevice);

	dim3 blockGrid(blockGridSize, blockGridSize);
	dim3 threadGrid(ThreadSize, ThreadSize);
	hipEventRecord(start2);
	for (int i = 0; i < size; i++)
	{
		SetCMatirxJordan << <blockGrid, threadGrid >> > (dMatrix, cMatrix, size, i);
		EleminatePartGPUJordan << <blockGrid, threadGrid >> > (dMatrix, cMatrix, dBVector, size, i);
		std::cout << "Aktualny krok: " << i << std::endl;
	}
	//printMatrix << <blockGrid, threadGrid >> > (dMatrix, size);
	DirectedOrderNumber *hostReulst;
	GetResovleVectorJordan << <blockGrid, threadGrid >> > (dMatrix, dBVector, dResultVector, size);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	float milisecondstwo = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventElapsedTime(&milisecondstwo, start2, stop);

	std::cout << "Czas w milisekundach po stronie GPU:" << milliseconds << std::endl;
	std::cout << "Czas w milisekundach po stronie GPU bez alkoacji pamieci:" << milisecondstwo << std::endl;
	hostReulst = new DirectedOrderNumber[size];
	hipMemcpy(hostReulst, dResultVector, resultVectorSize, hipMemcpyDeviceToHost);

	hipFree(dMatrix);
	hipFree(dResultVector);
	hipFree(cMatrix);
	hipFree(dBVector);

	return hostReulst;
}


DirectedOrderNumber** SimpleSampleEquationFourRowsTest()
{

	DirectedOrderNumber** sample;
	sample = new DirectedOrderNumber * [4];

	for (int i = 0; i < 4; i++)
		sample[i] = new DirectedOrderNumber[5];

	sample[0][0].SetValues(2.01, 5.99);
	sample[0][1].SetValues(-3.01, -0.99);
	sample[0][2].SetValues(1.99, 4.01);
	sample[0][3].SetValues(-0.99, -3.01);
	sample[0][4].SetValues(4.01, 7.99);

	sample[1][0].SetValues(1.99, 4.01);
	sample[1][1].SetValues(0.99, 1.01);
	sample[1][2].SetValues(2.99, 3.01);
	sample[1][3].SetValues(0.99, 3.01);
	sample[1][4].SetValues(4.01, 9.99);

	sample[2][0].SetValues(1.01, 2.99);
	sample[2][1].SetValues(2.01, 3.99);
	sample[2][2].SetValues(1.99, 2.01);
	sample[2][3].SetValues(0.99, 1.01);
	sample[2][4].SetValues(7.99, 12.01);

	sample[3][0].SetValues(1.99, 2.01);
	sample[3][1].SetValues(-2.01, -1.99);
	sample[3][2].SetValues(2.99, 3.01);
	sample[3][3].SetValues(1.99, 2.01);
	sample[3][4].SetValues(2.01, 1.99);

	return sample;
}




class FileHelper
{

public:
	double precsiosn;
	FileHelper()
	{
		precsiosn = 0.00000001;
	}
	FileHelper(int prec)
	{
		precsiosn = prec;
	}
	std::string GetPathToSampleFolder()
	{
		char* buffer;

		if ((buffer = getcwd(NULL, 0)) == NULL) {
			perror("failed to get current directory\n");
		}
		else {

			std::string pathToCurrentDirectory(buffer);
			pathToCurrentDirectory += "\\Sample";
			free(buffer);
			return pathToCurrentDirectory;
		}
		return "";


	}
	bool CheckDirectoryExist()
	{
		std::string path = GetPathToSampleFolder();
		std::ifstream file(path.c_str());
		bool isTrue = file.good();
		return isTrue;

	}

	int  CreateSampleFolder()
	{
		std::string pathToFolder = "md " + GetPathToSampleFolder();
		int x = system(pathToFolder.c_str());
		return x;
	}

	bool ChcekEquationFolderExisst(std::string folderName)
	{
		std::string path = GetPathToSampleFolder() + "\\" + folderName;
		std::ifstream file(path.c_str());
		bool isTrue = file.good();
		return isTrue;
	}

	int CreateEquationFolder(std::string folderName)
	{
		if (ChcekEquationFolderExisst(folderName))
			return 0;
		std::string pathToFolder = "md " + GetPathToSampleFolder() + "\\" + folderName;
		int x = system(pathToFolder.c_str());
		return x;
	}



	void SaveEquation(DirectedOrderNumber** equation, int rows, std::string fileName, std::string folderName)
	{

		std::string pathToFile = GetPathToSampleFolder();
		std::ofstream myfile;
		try {

			myfile.open(pathToFile + "\\" + folderName + "\\" + fileName + ".txt");
			myfile << rows << "\n";
			for (int i = 0; i < rows; i++)
			{
				for (int j = 0; j < rows + 1; j++)
				{
					myfile << "(" << std::fixed << std::setprecision(precsiosn) << (double)equation[i][j].infimum << "," << std::fixed << std::setprecision(precsiosn) << (double)equation[i][j].supremum << ")";

					myfile << ";";
				}
				myfile << "\n";

			}

			myfile.close();
		}
		catch (std::ifstream::failure e)
		{
			printf("Exception");
		}
	}

	void SaveVectorToFile(DirectedOrderNumber* frevaluesVector, std::string fileName, std::string folderName, int rows)
	{
		std::string pathToFile = GetPathToSampleFolder() + "\\" + folderName + "\\" + fileName + ".txt";
		std::ofstream myfile;

		try {

			myfile.open(pathToFile);

			for (int i = 0; i < rows; i++)
			{

				myfile << "(" << std::fixed << std::setprecision(precsiosn) << frevaluesVector[i].infimum << "," << std::fixed << std::setprecision(precsiosn) << frevaluesVector[i].supremum << ")";
				myfile << "\n";

			}

			myfile.close();
		}
		catch (std::ifstream::failure e)
		{
			printf("Exception");
		}
	}

	DirectedOrderNumber* ReadVectorFromFile(std::string fileName, std::string folderName, int rows)
	{
		std::string pathToFile = GetPathToSampleFolder() + "\\" + folderName + "\\" + fileName + ".txt";
		std::ifstream file;

		DirectedOrderNumber* vector;


		file.open(pathToFile.c_str());
		if (!file) {
			printf("Cannot read file");
			return NULL;
		}
		else
		{
			int counter = 0;
			std::string line;
			vector = new DirectedOrderNumber[rows];
			while (std::getline(file, line, '\n'))
			{
				vector[counter] = GetDirectedOrderNumberFromString(line);
				counter++;
			}
			return vector;
		}
		return NULL;
	}

	bool FileExist(std::string filename, std::string folderName)
	{
		std::string pathToFile = GetPathToSampleFolder() + "\\" + folderName + "\\" + filename + ".txt";
		std::ifstream file(pathToFile.c_str());
		bool isTrue = file.good();
		return isTrue;
	}


	DirectedOrderNumber** ReadEquation(std::string filename, std::string folderName, int* rowstoSave)
	{
		std::string pathToFile = GetPathToSampleFolder() + "\\" + folderName + "\\" + filename + ".txt";
		std::ifstream file;

		DirectedOrderNumber** equation;

		file.open(pathToFile.c_str());
		if (!file) {
			printf("Cannot read file");
			equation = CreateMatrix(1);
		}
		else
		{
			std::string line;
			std::getline(file, line);
			*rowstoSave = std::atoi(line.c_str());
			//	rowstoSave = &rows;
			equation = CreateMatrix(*rowstoSave);


			line = "";
			int count = 0;
			std::vector<std::string>* splittedStrings;
			while (std::getline(file, line, '\n'))
			{
				splittedStrings = StringRowToVector(line);

				SetNumberRowFromText(splittedStrings, equation, *rowstoSave, count);
				delete(splittedStrings);
				count++;
			}
			int x = 5;
		}
		file.close();
		return equation;
	}

	std::vector<std::string>* StringRowToVector(std::string row)
	{
		std::stringstream ss(row);
		std::string item;
		std::vector<std::string>* splittedStrings;
		splittedStrings = new std::vector<std::string>();

		while (std::getline(ss, item, ';'))
		{
			splittedStrings->push_back(item);
		}

		return splittedStrings;
	}

	void SetNumberRowFromText(std::vector<std::string>* row, DirectedOrderNumber** equation, int rows, int counter)
	{

		DirectedOrderNumber* number;
		number = NULL;
		std::string numberString;

		for (int i = 0; i < rows + 1; i++)
		{
			number = GetDirectedOrderNumberFromString(row->at(i));
			equation[counter][i].infimum = number->infimum;
			equation[counter][i].supremum = number->supremum;
		}
		delete(number);

	}

	DirectedOrderNumber* GetDirectedOrderNumberFromString(std::string numberInString)
	{
		DirectedOrderNumber* number;

		number = new DirectedOrderNumber(0.0, 0.0);

		numberInString.erase(std::remove(numberInString.begin(), numberInString.end(), '('), numberInString.end());
		numberInString.erase(std::remove(numberInString.begin(), numberInString.end(), ')'), numberInString.end());

		std::stringstream ss(numberInString);
		std::string line;
		std::vector<std::string>* splittedStrings;
		splittedStrings = new std::vector<std::string>();

		while (std::getline(ss, line, ','))
		{
			splittedStrings->push_back(line);
		}

		number->SetValues(std::stod(splittedStrings->at(0), NULL), std::stod(splittedStrings->at(1), NULL));
		return number;
	}

	DirectedOrderNumber** CreateMatrix(int rows)
	{
		DirectedOrderNumber** equation;
		equation = new  DirectedOrderNumber * [rows];

		for (int i = 0; i < rows; i++)
		{
			equation[i] = new DirectedOrderNumber[rows + 1];
		}

		return equation;
	}

};

class DirectedOrderNumberGenerator
{
private:

	double GenerateNumber()
	{

		int floor = (int)lowerRange;
		int roof = (int)upperRange;

		double result = (double)(rand() % roof + 1.0);
		return result;
	}

	double CreateSampleInfimum()
	{
		return (GenerateNumber() - precision);
	}

	double CreateSampleSupremum()
	{
		return (GenerateNumber() + precision);
	}

	DirectedOrderNumber GenerateDirectedOrderNumber()
	{
		double generatedNumber = GenerateNumber();

		if (rand() % 100 + 1 % 2 != 0) {
			generatedNumber *= -1.0;
		}

		double halfNumber = generatedNumber / 2.0;
		DirectedOrderNumber result;

		if (rand() % 100 + 1 % 2 == 0)
			result.SetValues(halfNumber - precision, halfNumber + precision);
		else
			result.SetValues(halfNumber + precision, halfNumber - precision);
		return result;
	}


public:
	double precision;
	int lowerRange;
	int upperRange;
	DirectedOrderNumberGenerator(int low, int up, double prec)
	{
		srand((unsigned)time(0));
		lowerRange = low;
		upperRange = up;
		precision = prec;

	}

	DirectedOrderNumber* GenerateSampleVector(int rows)
	{
		DirectedOrderNumber* resultVector;
		resultVector = new DirectedOrderNumber[rows];
		for (int i = 0; i < rows; i++)
		{
			resultVector[i] = GenerateDirectedOrderNumber();
		}

		return resultVector;
	}

	DirectedOrderNumber** GenerateEquationwithoutFreeValues(int rows)
	{
		DirectedOrderNumber** equation;
		equation = new DirectedOrderNumber * [rows];

		for (int i = 0; i < rows; i++)
			equation[i] = new DirectedOrderNumber[rows + 1];

		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows + 1; j++)
				equation[i][j] = GenerateDirectedOrderNumber();
			equation[i][rows] = GenerateDirectedOrderNumber();

		}

		return equation;
	}

	void GenereteFreeVaule(DirectedOrderNumber* equation, DirectedOrderNumber* resultVector, int rows)
	{

		DirectedOrderNumber tmp(0.0, 0.0);
		DirectedOrderNumber tmp2(0.0, 0.0);
		

		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < rows; j++)
			{
				tmp = GenerateDirectedOrderNumber();
				equation[i*rows+j].SetValues(tmp.infimum, tmp.supremum);
			}
			
			tmp2 = GenerateDirectedOrderNumber();
			resultVector[i].SetValues(tmp2.infimum, tmp2.supremum);
		
		}

	}


	void FillEquationWithFreeValues(DirectedOrderNumber** equation, DirectedOrderNumber* resultVector, int rows)
	{
		DirectedOrderNumber tmp(0.0, 0.0);
		//DirectedOrderNumber tmp2(0.0, 0.0);


		for (int i = 0; i < rows; i++)
		{	/*
			for (int j = 0; j < rows; j++)
			{
				tmp2 = operations.MultipleDirectedOrderNumber(equation[i][j], resultVector[i]);
				tmp = operations.AddDirectedOrderNumber(tmp, tmp2);
			}
			*/
			tmp = GenerateDirectedOrderNumber();
			equation[i][rows].SetValues(tmp.infimum, tmp.supremum);
		}
	}
};


std::string OptionList()
{

	std::string list = "1. Generete Equations Sample.\n";
	list += "2. Gauss Elimination.\n";
	list += "3. Gauss Jordan Elemination.\n";
	list += "4. Zaladuj uklad rownan z folderu Sample\n";
	return list;
}

int main()
{
	std::chrono::steady_clock::time_point start, end;
	std::chrono::duration<double> elapsed;

	int rows = 0;
	int chose = 100;
	int blocks = 0;
	int threads = 0;
	DirectedOrderNumberGenerator generator(1, 10, 0.00000001);


	DirectedOrderNumber* matrixToVector;
	DirectedOrderNumber* bVector;

	std::string fileName, folderName; 

	std::cout << "Wybierz opcje\n";

	while (chose != 0)
	{
		std::cout << OptionList();
		std::cin >> chose;
		
		switch (chose)
		{
		case 1:
			std::cout << "podaj wielkosc ukladu rownan\n";
			std::cin >> rows;

	

			matrixToVector = new DirectedOrderNumber[rows * rows];
			bVector = new DirectedOrderNumber[rows];
			generator.GenereteFreeVaule(matrixToVector, bVector, rows);
			/*for (int i = 0; i < rows; i++)
			{
				for (int j = 0; j < rows; j++)
				{
					std::cout << "liczba" << matrixToVector[i * rows + j].infimum << " ";
				}
				std::cout << std::endl;
			}
			*/
			break;

		case 2:
			std::cout << "wielkosc ukladu: " << rows << std::endl;
			std::cout << "Podaj ilosc blokow" << std::endl;
			std::cin >> blocks;
			std::cout << "Podaj ilosc w�ktk�w w bloku" << std::endl;
			std::cin >> threads;

			std::cout << "Uruchiomienie algorytmu eliminacji Gaussa\n";
			start = std::chrono::high_resolution_clock::now();
			GaussElemination(matrixToVector, bVector, rows, blocks, threads);
			end = std::chrono::high_resolution_clock::now();
			std::cout << "Czas trwania: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "\n";
			break;

		case 3:
			std::cout << "wielkosc ukladu: " << rows << std::endl;
			std::cout << "Podaj ilosc blokow" << std::endl;
			std::cin >> blocks;
			std::cout << "Podaj ilosc w�ktk�w w bloku" << std::endl;
			std::cin >> threads;

			std::cout << "Uruchiomienie algorytmu eliminacji Gaussa-Jordana\n";
			start = std::chrono::high_resolution_clock::now();
			//GaussEleminationGpu(matrixToVector, bVector, rows, blocks, threads);
			GaussJordanElemination(matrixToVector, bVector, rows, blocks, threads);
			end = std::chrono::high_resolution_clock::now();

			std::cout << "Czas trwania: " << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() << "\n";
			break;

		case 4:
			break;
		}
	}
	delete[] bVector;
	return 0;
}

